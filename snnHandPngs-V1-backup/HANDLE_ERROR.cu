/*

 * HANDLE_ERROR.cu
 *
 *  Created on: Aug 5, 2015
 *      Author: banafsheh
 */
#include <stdio.h>
#include <iostream>
#include "HANDLE_ERROR.h"

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),
				file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


